#include "hip/hip_runtime.h"
#include "noarr-structures/include/noarr/structures.hpp"
#include "noarr-structures/include/noarr/structures_extended.hpp"
#include "noarr-structures/include/noarr/structures/shortcuts.hpp"
#include "noarr-structures/include/noarr/structures/traverser.hpp"

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>

using data_t = float;

using namespace noarr::literals;

char *volatile volatile_shower;

template <typename layout_t>
auto my_make_bag(layout_t layout)
{
    auto size = layout | noarr::get_size();
    char *ptr = new char[size];
    volatile_shower = ptr;

    char *tmp = volatile_shower;
    std::unique_ptr<char[]> unique(tmp);

    return std::make_tuple(noarr::make_bag(layout, tmp), std::move(unique));
}

template <typename bag_t>
void fill_matrix(bag_t &bag)
{
    // std::mt19937 gen(0);
    // std::uniform_real_distribution<> dis(1.0, 2.0);
    size_t i = 0;
    noarr::traverser(bag.structure().unwrap()).for_each([&](auto idx)
                                                        { bag.at(idx) = (data_t)i++; });
}

template <typename bag_t>
void fill_matrix_zero(bag_t &bag)
{
    noarr::traverser(bag.structure().unwrap()).for_each([&](auto idx)
                                                        { bag.at(idx) = (data_t)0; });
}

template <typename bagl_t, typename bagr_t>
void compare(bagl_t &bagl, bagr_t &bagr)
{
    noarr::traverser(bagl.structure().unwrap(), bagr.structure().unwrap()).for_each([&](auto idxl, auto idxr)
                                                                                    {
        if (bagl.at(idxl) != bagr.at(idxr))
            std::cout << bagl.at(idxl) << "!=" << bagr.at(idxr) << std::endl; });
}

template <typename A_t>
__global__ void hadamard_hand(size_t I, size_t J, A_t A, const char *__restrict__ a, const char *__restrict__ b, char *__restrict__ c)
{
    auto AA = A ^ noarr::fix<'n'>(threadIdx.x / warpSize);

    for (size_t aa = threadIdx.x % warpSize; aa < I * J; aa += warpSize)
    {
        size_t i = aa / J, j = aa % J;
        (AA | noarr::get_at<'i', 'j'>(c, i, j)) = (AA | noarr::get_at<'i', 'j'>(a, i, j)) * (AA | noarr::get_at<'i', 'j'>(b, i, j));
    }
}

template <typename A_t>
__global__ void hadamard_noarr(A_t A, const char *__restrict__ a, const char *__restrict__ b, char *__restrict__ c)
{
    auto AA = A ^ noarr::fix<'n'>(threadIdx.x / warpSize);

    noarr::traverser(AA).order(noarr::merge_blocks<'i', 'j', 'a'>() ^
                               noarr::step<'a'>(threadIdx.x % warpSize, warpSize) ^
                               noarr::reorder<'a'>())
        .for_each([=](auto idx)
                  {
                    //   const float2 a2 = reinterpret_cast<const float2&>(AA | noarr::get_at(a, idx));
                    //   const float2 b2 = reinterpret_cast<const float2&>(AA | noarr::get_at(b, idx));
                    //   reinterpret_cast<float2&>(AA | noarr::get_at(c, idx)) = {a2.x*b2.x, a2.y*b2.y};
            (AA | noarr::get_at(c, idx)) = (AA | noarr::get_at(a, idx)) * (AA | noarr::get_at(b, idx)); });
}

void hadamard(size_t I, size_t J, size_t iters)
{
    auto A = noarr::array<'n', 16, noarr::vector<'i', noarr::vector<'j', noarr::scalar<data_t>>>>() ^ noarr::set_length<'i'>(I) ^ noarr::set_length<'j'>(J);

    auto [c_bag, cu] = my_make_bag(A);
    auto [a_bag, au] = my_make_bag(A);
    auto [b_bag, bu] = my_make_bag(A);

    fill_matrix(a_bag);
    fill_matrix(b_bag);

    char *a_data;
    char *b_data;
    char *c_data;
    hipMalloc(&a_data, I * J * 16 * sizeof(data_t));
    hipMalloc(&b_data, I * J * 16 * sizeof(data_t));
    hipMalloc(&c_data, I * J * 16 * sizeof(data_t));

    hipMemcpy(a_data, a_bag.data(), I * J * 16 * sizeof(data_t), hipMemcpyHostToDevice);
    hipMemcpy(b_data, b_bag.data(), I * J * 16 * sizeof(data_t), hipMemcpyHostToDevice);
    hipMemcpy(c_data, c_bag.data(), I * J * 16 * sizeof(data_t), hipMemcpyHostToDevice);

    for (size_t i = 0; i < iters; i++)
    {
        fill_matrix_zero(c_bag);

        auto start = std::chrono::steady_clock::now();

        hadamard_noarr<<<1, 512>>>(A, a_data, b_data, c_data);
        hipDeviceSynchronize();

        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsed_seconds = end - start;

        hipMemcpy(c_bag.data(), c_data, I * J * 16 * sizeof(data_t), hipMemcpyDeviceToHost);

        std::cout << "hadamard,traverser," << elapsed_seconds.count() << std::endl;
    }

    auto [c_bag2, cu2] = my_make_bag(A);
    for (size_t i = 0; i < iters; i++)
    {
        fill_matrix_zero(c_bag2);

        auto start = std::chrono::steady_clock::now();

        hadamard_hand<<<1, 512>>>(I, J, A, a_data, b_data, c_data);
        hipDeviceSynchronize();

        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsed_seconds = end - start;

        hipMemcpy(c_bag2.data(), c_data, I * J * 16 * sizeof(data_t), hipMemcpyDeviceToHost);

        std::cout << "hadamard,hand," << elapsed_seconds.count() << std::endl;
    }

    hipFree(a_data);
    hipFree(b_data);
    hipFree(c_data);

    compare(c_bag, c_bag2);
}

int main(int argc, char **argv)
{
    std::vector<std::string> args(argv + 1, argv + argc);

    size_t I = std::stoi(args[0]);
    size_t J = std::stoi(args[1]);

    size_t iters = 30;

    hadamard(I, J, iters);

    return 0;
}
